
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>

#define Inf 9999
#define inf 9999
#define INF 9999

float *dist;
float *kernel1;
float *kernel2;
float *kernel3;
struct timeval startwtime, endwtime;
double seq_time;

__global__ void floydWarshellKernel2 (float* dist, int k, int n);
__global__ void floydWarshellKernel1(float *dist, int k, int n);
__global__ void floydWarshellKernel3(float *dist, int k, int n);
void floydWarshellSerial (float* graph, float* result, int n);
void printSolution(float* dist, int n);
void Check(float* array1, float* array2, int n);

int main(int argc, char*argv[]){


  int n;

/*Check Arguments*/
  if (argc!=3){
    printf("Error, two arguments are needed. arg1 =  full path of the input txt file"
    " which contains the matrix, arg2 = n, where n X n is matrix dimension \n");
    exit(1);
  }

/*Open file*/
  FILE *inputMatrix;
  inputMatrix=fopen(argv[1], "r+");

/*Check if success*/
  if (inputMatrix==NULL){
    printf("Error opening file. Check file permissions\n");
    exit(1);
  }

  n= 1<<atoi(argv[2]);

  float *graph;
  graph=(float*)malloc(n*n*sizeof(float));

  //printf("Initial Distance-Matrix between vertices is:\n");

  for (int i=0; i<n; i++){
    for (int j=0; j<n; j++){
      fscanf(inputMatrix, "%f", &graph[j+i*n]);
    }
  }
  printf("\n");
  fclose(inputMatrix);

  kernel1=(float*)malloc(n*n*sizeof(float));
  kernel2=(float*)malloc(n*n*sizeof(float));
  kernel3=(float*)malloc(n*n*sizeof(float));

  //////////////////Serial algorithm/////////////////////

  gettimeofday (&startwtime, NULL);
  float *result;
  result=(float*)malloc(n*n*sizeof(float));

  floydWarshellSerial(graph, result, n);

  gettimeofday (&endwtime, NULL);

  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
                        + endwtime.tv_sec - startwtime.tv_sec);

  printf("Serial time: %f\n", seq_time);

  ///////////////Cuda kernel 1 algorithm///////////////////


  int blocksize= 4;
  dim3 dimBlock( blocksize, blocksize );
  dim3 dimGrid( n/dimBlock.x, n/dimBlock.y );

  gettimeofday (&startwtime, NULL);
  hipMalloc((void**)&dist, n*n*sizeof(float));
  hipMemcpy(dist, graph, n*n*sizeof(float), hipMemcpyHostToDevice);
  for (int k=0; k<n; k++) floydWarshellKernel1<<<dimGrid, dimBlock>>>(dist, k, n);
  hipMemcpy(kernel1, dist, n*n*sizeof(float), hipMemcpyDeviceToHost);
  gettimeofday (&endwtime, NULL);
  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
                        + endwtime.tv_sec - startwtime.tv_sec);

  Check(kernel1, result, n);
  printf("Cuda time, kernel 1: %f \n", seq_time);

  ///////////////Cuda kernel 2 algorithm///////////////////
  blocksize= 4;
  dim3 dimBlock2( blocksize, blocksize );
  dim3 dimGrid2(n, (n+blocksize-1)/blocksize);

  gettimeofday (&startwtime, NULL);
  hipMemcpy(dist, graph, n*n*sizeof(float), hipMemcpyHostToDevice);
  for (int k=0; k<n; k++) {
    floydWarshellKernel2<<<dimGrid2, dimBlock2>>>(dist, k, n);
  }
  hipMemcpy(kernel2, dist, n*n*sizeof(float), hipMemcpyDeviceToHost);
  gettimeofday (&endwtime, NULL);
  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
                        + endwtime.tv_sec - startwtime.tv_sec);

  Check(kernel2, result, n);
  printf("Cuda time, kernel 2: %f\n", seq_time);

  ///////////////Cuda kernel 3 algorithm///////////////////


  blocksize= 128;
  int gridsize=n/blocksize;

  gettimeofday (&startwtime, NULL);
  hipMalloc((void**)&dist, n*n*sizeof(float));
  hipMemcpy(dist, graph, n*n*sizeof(float), hipMemcpyHostToDevice);
  for (int k=0; k<n; k++) {
    floydWarshellKernel3<<<gridsize, blocksize>>>(dist, k, n);
  }
  hipMemcpy(kernel3, dist, n*n*sizeof(float), hipMemcpyDeviceToHost);
  gettimeofday (&endwtime, NULL);
  seq_time = (double)((endwtime.tv_usec - startwtime.tv_usec)/1.0e6
                        + endwtime.tv_sec - startwtime.tv_sec);

  Check(kernel3, result, n);
  printf("Cuda time, kernel 3: %f\n", seq_time);

}

__global__ void floydWarshellKernel2(float *dist, int k, int n)
{

  int j=blockIdx.y*blockDim.y + threadIdx.y;
	if(j>=n) return;
	int idx=n*blockIdx.x+j;

	__shared__ float best;

	if(threadIdx.y==0) best=dist[n*blockIdx.x+k];
	__syncthreads();

	if(dist[k*n+j]+best<dist[idx]){
		dist[idx]=dist[k*n+j]+best;
	}
}

__global__ void floydWarshellKernel1(float *dist, int k, int n)
{


    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i*n + j;

    if (i<n && j<n){
      if (dist[k+i*n] + dist[j+k*n] < dist[index]){
        dist[index] = dist[k+i*n]+dist[j+k*n];
      }
    }
    __syncthreads();

}

void floydWarshellSerial(float *graph, float *result, int n)
{
    for (int i = 0; i<n; i++){
      for (int j = 0; j<n; j++){
        result[j+i*n] = graph[j+i*n];
      }
    }

    for (int k=0; k<n; k++)
    {
        // Pick all vertices as source one by one
        for (int i=0; i<n; i++)
        {
            // Pick all vertices as destination for the
            // above picked source
            for (int j=0; j<n; j++)
            {

                // If vertex k is on the shortest path from
                // i to j, then update the value of Distance-Matrix[i][j]
                if (result[k+i*n] + result[j+k*n] < result[j+i*n])
                    result[j+i*n] = result[k+i*n]+result[j+k*n];
            }
        }
    }
}

__global__ void floydWarshellKernel3(float *dist, int k, int n)
{

    // int i=blockIdx.x * blockDim.x + threadIdx.x;
    // int index=0;
    // for (int j=0; j<n; j++){
    //   if(i<n){
    //     index=i*n+j;
    //     if (dist[k+i*n] + dist[j+k*n] < dist[index]){
    //       dist[index] = dist[k+i*n]+dist[j+k*n];
    //     }
    //   }
    // }
    int j= blockDim.x*blockIdx.x+threadIdx.x;
    if (j>=n) return;

    __shared__ float best;

    for (int i=0; i<n; i++){
      if (threadIdx.x==0) best=dist[n*i+k];
      __syncthreads();

      if(best+dist[k*n+j]<dist[n*i+j]){
        dist[n*i+j]=best+dist[k*n+j];
      }
    }

}

void printSolution(float* dist, int n){
  printf ("Following matrix shows the shortest distances"
            " between every pair of vertices \n");
  printf("\n");
    for (int i=0; i<n; i++)
    {
        for (int j=0; j<n; j++)
        {
                printf ("%f ", dist[j+i*n]);
        }
        printf("\n");
    }
}

void Check(float* array1, float* array2, int n){
  for (int i=0; i<n*n; i++){
    if (array1[i]!=array2[i]){
      printf("Incorrect Solution\n");
      exit(1);
    }
  }
  printf("Correct Solution\n");
}
